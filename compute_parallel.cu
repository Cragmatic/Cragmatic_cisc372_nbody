#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <math.h>
#include "vector.h"
#include "config.h"
#include "stdio.h"

//My Kernel
//Whatever I called it
__global__ void pairwise_accel(vector3** accels, vector3* hPos, vector3* hVel, double* mass) {
	int k;
	//Assuming we spawn enough blocks+threads to cover the whole NUMENTITIESxNUMENTITIES matrix, each thread does 1 calculation
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	/*
	if (j == 1000) {
		printf("Hello from thread coordinates %d, %d with args bidx.x: %d, bdim.x: %d, tidx.x: %d, bidx.y: %d, bdim.y: %d, tidx.y: %d, \n", 
	i, j, blockIdx.x, blockDim.x, threadIdx.x, blockIdx.y, blockDim.y, threadIdx.y);
	}
	*/
	if (i > NUMENTITIES || j > NUMENTITIES) {
		return;
	}
	if (i==j) {
		FILL_VECTOR(accels[i][j],0,0,0);
	}
	else{
		vector3 distance;
		for (k=0;k<3;k++) distance[k]=hPos[i][k]-hPos[j][k];
		double magnitude_sq=distance[0]*distance[0]+distance[1]*distance[1]+distance[2]*distance[2];
		double magnitude=sqrt(magnitude_sq);
		double accelmag=-1*GRAV_CONSTANT*mass[j]/magnitude_sq;
		FILL_VECTOR(accels[i][j],accelmag*distance[0]/magnitude,accelmag*distance[1]/magnitude,accelmag*distance[2]/magnitude);
	}
	__syncthreads();

	//sum up the rows of our matrix to get effect on each entity, then update velocity and position.
		vector3 accel_sum={0,0,0};
		for (k=0;k<3;k++)
			accel_sum[k]+=accels[i][j][k];
		
//????
	__syncthreads();

		//compute the new velocity based on the acceleration and time interval
		//compute the new position based on the velocity and time interval
		for (k=0;k<3;k++){
			hVel[i][k]+=accel_sum[k]*INTERVAL;
			hPos[i][k]=hVel[i][k]*INTERVAL;
		}
}


//compute: Updates the positions and locations of the objects in the system based on gravity.
//Parameters: None
//Returns: None
//Side Effect: Modifies the hPos and hVel arrays with the new positions and accelerations after 1 INTERVAL
void compute(vector3* d_hPos, vector3* d_hVel, dim3 dimBlock, dim3 dimGrid){
	//make an acceleration matrix which is NUMENTITIES squared in size;
	int i,j,k;
	vector3* values=(vector3*)malloc(sizeof(vector3)*NUMENTITIES*NUMENTITIES);
	vector3** accels=(vector3**)malloc(sizeof(vector3*)*NUMENTITIES);
	for (i=0;i<NUMENTITIES;i++)
		accels[i]=&values[i*NUMENTITIES];
	/**
	//first compute the pairwise accelerations.  Effect is on the first argument.
	for (i=0;i<NUMENTITIES;i++){
		for (j=0;j<NUMENTITIES;j++){
			if (i==j) {
				FILL_VECTOR(accels[i][j],0,0,0);
			}
			else{
				vector3 distance;
				for (k=0;k<3;k++) distance[k]=hPos[i][k]-hPos[j][k];
				double magnitude_sq=distance[0]*distance[0]+distance[1]*distance[1]+distance[2]*distance[2];
				double magnitude=sqrt(magnitude_sq);
				double accelmag=-1*GRAV_CONSTANT*mass[j]/magnitude_sq;
				FILL_VECTOR(accels[i][j],accelmag*distance[0]/magnitude,accelmag*distance[1]/magnitude,accelmag*distance[2]/magnitude);
			}
		}
	}
	Commented out original code */


	//MY CODE SECTION (1st attempt):
	vector3** d_accels;
	vector3* d_values
	hipMalloc(&d_values, sizeof(vector3)*NUMENTITIES*NUMENTITIES);
	hipMalloc(&d_accels, sizeof(vector3*)*NUMENTITIES);
	pairwise_accel<<<dimGrid, dimBlock>>>(d_accels, d_hPos, d_hVel, mass);
	//END MY CODE SECTION


	/*COMMENTED OUT
	//sum up the rows of our matrix to get effect on each entity, then update velocity and position.
	for (i=0;i<NUMENTITIES;i++){
		vector3 accel_sum={0,0,0};
		for (j=0;j<NUMENTITIES;j++){
			for (k=0;k<3;k++)
				accel_sum[k]+=accels[i][j][k];
		}
		//compute the new velocity based on the acceleration and time interval
		//compute the new position based on the velocity and time interval
		for (k=0;k<3;k++){
			hVel[i][k]+=accel_sum[k]*INTERVAL;
			hPos[i][k]=hVel[i][k]*INTERVAL;
		}
	}
	*/
	free(accels);
	free(values);

	//Parallel Frees
	hipFree(d_accels);
}